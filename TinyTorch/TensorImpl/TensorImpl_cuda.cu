#include "hip/hip_runtime.h"
/*
 * TinyTorch
 * @author 	: keith@robot9.me
 *
 */

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include <cassert>
#include <cfloat>
#include <iostream>

#include "TensorImpl_cpu.h"
#include "TensorImpl_cuda.cuh"
#include "TensorImpl_cuda.inc"

namespace TinyTorch {

const char* curandGetErrorString(hiprandStatus_t status);
const char* cublasGetErrorString(hipblasStatus_t status);

#define CUDA_CHECK(call)                                                      \
  do {                                                                        \
    hipError_t err = call;                                                   \
    if (err != hipSuccess) {                                                 \
      std::cerr << "CUDA error in file '" << __FILE__ << "' in line "         \
                << __LINE__ << ": " << hipGetErrorString(err) << " (" << err \
                << ")" << std::endl;                                          \
      abort();                                                                \
    }                                                                         \
  } while (0)

#define CURAND_CHECK(call)                                               \
  do {                                                                   \
    hiprandStatus_t err = call;                                           \
    if (err != HIPRAND_STATUS_SUCCESS) {                                  \
      std::cerr << "CURAND error in file '" << __FILE__ << "' in line "  \
                << __LINE__ << ": " << curandGetErrorString(err) << " (" \
                << err << ")" << std::endl;                              \
      abort();                                                           \
    }                                                                    \
  } while (0)

#define CUBLAS_CHECK(call)                                               \
  do {                                                                   \
    hipblasStatus_t err = call;                                           \
    if (err != HIPBLAS_STATUS_SUCCESS) {                                  \
      std::cerr << "CUBLAS error in file '" << __FILE__ << "' in line "  \
                << __LINE__ << ": " << cublasGetErrorString(err) << " (" \
                << err << ")" << std::endl;                              \
      abort();                                                           \
    }                                                                    \
  } while (0)

#define CUDA_KERNEL_CHECK()                                                   \
  do {                                                                        \
    hipError_t err = hipGetLastError();                                     \
    if (err != hipSuccess) {                                                 \
      std::cerr << "CUDA kernel error in file '" << __FILE__ << "' in line "  \
                << __LINE__ << ": " << hipGetErrorString(err) << " (" << err \
                << ")" << std::endl;                                          \
      abort();                                                                \
    }                                                                         \
  } while (0)

static std::random_device _r;
unsigned long RandomGeneratorCUDA::seed_ = _r();
unsigned long RandomGeneratorCUDA::sequence_ = 0;

void* AllocatorCPU::allocatePinned(size_t size) {
  void* ptr = nullptr;
  CUDA_CHECK(hipHostMalloc(&ptr, size));
  return ptr;
}

void AllocatorCPU::deallocatePinned(void* ptr) {
  CUDA_CHECK(hipHostFree(ptr));
}

void AllocatorCUDA::allocate(void** ptr, size_t size) {
  CUDA_CHECK(hipMalloc(ptr, size));
}

void AllocatorCUDA::deallocate(void* ptr) {
  if (ptr) {
    CUDA_CHECK(hipFree(ptr));
  }
}

TensorOpsCUDA::TensorOpsCUDA(int32_t device, size_t blockSize)
    : cudaDeviceIdx_(device), blockSize_(blockSize) {
  CUDA_CHECK(hipSetDevice(cudaDeviceIdx_));
  CUDA_CHECK(hipGetDeviceProperties(&deviceProp_, cudaDeviceIdx_));

  if (blockSize_ > deviceProp_.maxThreadsPerBlock) {
    blockSize_ = deviceProp_.maxThreadsPerBlock;
  }
}

TensorOpsCUDA::~TensorOpsCUDA() {
  allocator_.clear();
  if (blasHandle_) {
    CUBLAS_CHECK(hipblasDestroy(blasHandle_));
  }
}

hipblasHandle_t TensorOpsCUDA::getCublasHandle() {
  if (blasHandle_ == nullptr) {
    CUBLAS_CHECK(hipblasCreate(&blasHandle_));
  }
  return blasHandle_;
}

TensorCudaCtx TensorOpsCUDA::getTensorCtx(const TensorImpl& t) {
  TensorCudaCtx ret{};
  ret.dimCount_ = t.dimCount_;
  ret.elemCount_ = t.elemCount_;
  memcpy(ret.shape_, t.shape_.data(), t.dimCount_ * sizeof(int32_t));
  memcpy(ret.strides_, t.strides_.data(), t.dimCount_ * sizeof(int32_t));
  ret.data_ = t.data_;
  return ret;
}

template <typename OP>
void TensorOpsCUDA::opSingle_(TensorImpl& t) const {
  kSingleOp_<OP>
      <<<getGridSize(t.elemCount_), getBlockSize()>>>(t.data_, t.elemCount_);
  CUDA_KERNEL_CHECK();
}

template <typename OP>
TensorImpl TensorOpsCUDA::opSingle(const TensorImpl& t) const {
  auto result = TensorImpl::shape(t.shape(), t.device_);
  kSingleOp<OP><<<getGridSize(t.elemCount_), getBlockSize()>>>(
      result.data_, t.data_, t.elemCount_);
  CUDA_KERNEL_CHECK();
  return result;
}

template <typename OP>
TensorImpl TensorOpsCUDA::opPair(const TensorImpl& a,
                                 const TensorImpl& b) const {
  auto result = TensorImpl::shape(a.shape(), a.device_);
  kPairOp<OP><<<getGridSize(result.elemCount_), getBlockSize()>>>(
      result.data_, a.data_, b.data_, result.elemCount_);
  CUDA_KERNEL_CHECK();
  return result;
}

template <typename OP>
TensorImpl TensorOpsCUDA::opPair(const TensorImpl& a, float b) const {
  auto result = TensorImpl::shape(a.shape(), a.device_);
  kPairScalarSecondOp<OP><<<getGridSize(a.elemCount_), getBlockSize()>>>(
      result.data_, a.data_, b, a.elemCount_);
  CUDA_KERNEL_CHECK();
  return result;
}

template <typename OP>
TensorImpl TensorOpsCUDA::opPair(float a, const TensorImpl& b) const {
  auto result = TensorImpl::shape(b.shape(), b.device_);
  kPairScalarFirstOp<OP><<<getGridSize(b.elemCount_), getBlockSize()>>>(
      result.data_, a, b.data_, b.elemCount_);
  CUDA_KERNEL_CHECK();
  return result;
}

template <typename OP>
TensorImpl TensorOpsCUDA::opPairScalarFirst(const TensorImpl& a,
                                            const TensorImpl& b) const {
  auto result = TensorImpl::shape(b.shape(), b.device_);
  kPairScalarFirstOp<OP><<<getGridSize(result.elemCount_), getBlockSize()>>>(
      result.data_, a.data_, b.data_, result.elemCount_);
  CUDA_KERNEL_CHECK();
  return result;
}

template <typename OP>
TensorImpl TensorOpsCUDA::opPairScalarSecond(const TensorImpl& a,
                                             const TensorImpl& b) const {
  auto result = TensorImpl::shape(a.shape(), a.device_);
  kPairScalarSecondOp<OP><<<getGridSize(result.elemCount_), getBlockSize()>>>(
      result.data_, a.data_, b.data_, result.elemCount_);
  CUDA_KERNEL_CHECK();
  return result;
}

template <typename OP>
void TensorOpsCUDA::opPair_(TensorImpl& t, float b) const {
  kPairScalarSecondOp_<OP>
      <<<getGridSize(t.elemCount_), getBlockSize()>>>(t.data_, b, t.elemCount_);
  CUDA_KERNEL_CHECK();
}

template <typename OP>
void TensorOpsCUDA::opPair_(TensorImpl& t, const TensorImpl& b) const {
  kPairOp_<OP><<<getGridSize(t.elemCount_), getBlockSize()>>>(t.data_, b.data_,
                                                              t.elemCount_);
  CUDA_KERNEL_CHECK();
}

template <typename OP>
void TensorOpsCUDA::opPairScalarFirst_(TensorImpl& a,
                                       const TensorImpl& b) const {
  auto result = TensorImpl::shape(b.shape_, b.device_);
  kPairScalarFirstOp<OP><<<getGridSize(result.elemCount_), getBlockSize()>>>(
      result.data_, a.data_, b.data_, result.elemCount_);
  CUDA_KERNEL_CHECK();
  a = std::move(result);
}

template <typename OP>
void TensorOpsCUDA::opPairScalarSecond_(TensorImpl& a,
                                        const TensorImpl& b) const {
  kPairScalarSecondOp_<OP><<<getGridSize(a.elemCount_), getBlockSize()>>>(
      a.data_, b.data_, a.elemCount_);
  CUDA_KERNEL_CHECK();
}

template <typename OP>
void TensorOpsCUDA::broadcastImpl(TensorImpl& result, const TensorImpl& a,
                                  const TensorImpl& b) const {
  // fast broadcast with a
  if (b.elemCount_ == result.elemCount_) {
    if (isLeadingOnes(a.shape())) {
      kBroadcastOpFast<OP, true, true>
          <<<getGridSize(result.elemCount_), getBlockSize()>>>(
              result.data_, a.data_, b.data_, a.elemCount_, result.elemCount_);
      CUDA_KERNEL_CHECK();
      return;
    }

    if (isTrailingOnes(a.shape())) {
      kBroadcastOpFast<OP, false, true>
          <<<getGridSize(result.elemCount_), getBlockSize()>>>(
              result.data_, a.data_, b.data_, result.elemCount_ / a.elemCount_,
              result.elemCount_);
      CUDA_KERNEL_CHECK();
      return;
    }
  }

  // fast broadcast with b
  if (a.elemCount_ == result.elemCount_) {
    if (isLeadingOnes(b.shape())) {
      kBroadcastOpFast<OP, true, false>
          <<<getGridSize(result.elemCount_), getBlockSize()>>>(
              result.data_, a.data_, b.data_, b.elemCount_, result.elemCount_);
      CUDA_KERNEL_CHECK();
      return;
    }

    if (isTrailingOnes(b.shape())) {
      kBroadcastOpFast<OP, false, false>
          <<<getGridSize(result.elemCount_), getBlockSize()>>>(
              result.data_, a.data_, b.data_, result.elemCount_ / b.elemCount_,
              result.elemCount_);
      CUDA_KERNEL_CHECK();
      return;
    }
  }

  const auto ctxA = getTensorCtx(a);
  const auto ctxB = getTensorCtx(b);
  const auto ctxC = getTensorCtx(result);
  kBroadcastOpCommon<OP><<<getGridSize(result.elemCount_), getBlockSize()>>>(
      ctxC, ctxA, ctxB, result.elemCount_);
  CUDA_KERNEL_CHECK();
}

template <typename OP>
TensorImpl TensorOpsCUDA::opPairBroadcast(const TensorImpl& a,
                                          const TensorImpl& b) const {
  Shape retShape;
  auto comp = checkShapeCompatible(a.shape(), b.shape(), retShape);
  if (comp == ShapeCompatible_Error) {
    error(__FUNCTION__, TensorError_ShapeNotAligned);
    return {};
  }

  if (comp == ShapeCompatible_SameShape) {
    return opPair<OP>(a, b);
  }

  auto result = TensorImpl::shape(retShape, a.device_);
  broadcastImpl<OP>(result, a, b);
  return result;
}

template <typename OP>
void TensorOpsCUDA::opPairBroadcast_(TensorImpl& a, const TensorImpl& b) const {
  Shape retShape;
  auto comp = checkShapeCompatible(a.shape(), b.shape(), retShape);
  if (comp == ShapeCompatible_Error) {
    error(__FUNCTION__, TensorError_ShapeNotAligned);
    return;
  }
  if (comp == ShapeCompatible_SameShape) {
    opPair_<OP>(a, b);
    return;
  }

  auto result = TensorImpl::shape(retShape, a.device_);
  broadcastImpl<OP>(result, a, b);
  a = std::move(result);
}

template <typename OP, typename IndexFunc>
void TensorOpsCUDA::reduceMerge(float* values, const float* input, int32_t n,
                                int32_t m) {
  auto blocks = getGridSize(n);

  float* dTmp = nullptr;
  allocate(reinterpret_cast<void**>(&dTmp), m * blocks * sizeof(float));

  kReduceMerge<OP, IndexFunc>
      <<<m * blocks, getBlockSize()>>>(dTmp, input, n, m);
  CUDA_KERNEL_CHECK();

  while (blocks > 1) {
    const auto currBlocks = blocks;
    blocks = getGridSize(currBlocks);
    kReduceMerge<OP, IndexFunc>
        <<<m * blocks, getBlockSize()>>>(dTmp, dTmp, currBlocks, m);
    CUDA_KERNEL_CHECK();
  }
  copyOnDevice(values, dTmp, m * sizeof(float));
  deallocate(dTmp);
}

template <typename OP, typename IndexFunc>
void TensorOpsCUDA::reduceIdxMerge(float* values, float* indices,
                                   const float* input, int32_t n, int32_t m) {
  auto blocks = getGridSize(n);

  float* tmpValues = nullptr;
  float* tmpIndices = nullptr;
  allocate(reinterpret_cast<void**>(&tmpValues), m * blocks * sizeof(float));
  allocate(reinterpret_cast<void**>(&tmpIndices), m * blocks * sizeof(float));

  kReduceIdxMerge<OP, IndexFunc><<<m * blocks, getBlockSize()>>>(
      tmpValues, tmpIndices, input, nullptr, n, m);
  CUDA_KERNEL_CHECK();

  while (blocks > 1) {
    const auto currBlocks = blocks;
    blocks = getGridSize(currBlocks);
    kReduceIdxMerge<OP, IndexFunc><<<m * blocks, getBlockSize()>>>(
        tmpValues, tmpIndices, tmpValues, tmpIndices, currBlocks, m);
    CUDA_KERNEL_CHECK();
  }
  if (values) {
    copyOnDevice(values, tmpValues, m * sizeof(float));
  }
  if (indices) {
    copyOnDevice(indices, tmpIndices, m * sizeof(float));
  }
  deallocate(tmpValues);
  deallocate(tmpIndices);
}

template <typename OP>
void TensorOpsCUDA::reduceDimFirst(float* values, const float* input, int32_t n,
                                   int32_t m) {
  auto tmp = TensorImpl::shape({m * n}, Device::CUDA);
  transpose2D(tmp.data_, input, m, n);
  reduceMerge<OP, ReduceIndexLastDim>(values, tmp.data_, n, m);

  // slower than transpose
  // reduceMerge<OP, ReduceIndexFirstDim>(values, input, n, m);
}

template <typename OP>
void TensorOpsCUDA::reduceDimLast(float* values, const float* input, int32_t n,
                                  int32_t m) {
  reduceMerge<OP, ReduceIndexLastDim>(values, input, n, m);
}

template <typename OP>
void TensorOpsCUDA::reduceIdxDimFirst(float* values, float* indices,
                                      const float* input, int32_t n,
                                      int32_t m) {
  auto tmp = TensorImpl::shape({m * n}, Device::CUDA);
  transpose2D(tmp.data_, input, m, n);
  reduceIdxMerge<OP, ReduceIndexLastDim>(values, indices, tmp.data_, n, m);

  // slower than transpose
  // reduceIdxMerge<OP, ReduceIndexFirstDim>(values, indices, input, n, m);
}

template <typename OP>
void TensorOpsCUDA::reduceIdxDimLast(float* values, float* indices,
                                     const float* input, int32_t n, int32_t m) {
  reduceIdxMerge<OP, ReduceIndexLastDim>(values, indices, input, n, m);
}

template <typename OP>
TensorImpl TensorOpsCUDA::reduceDim(const TensorImpl& t, int32_t dim,
                                    bool keepDims) {
  if (dim < 0) {
    dim += t.dimCount_;
  }
  if (dim < 0 || dim >= t.dimCount_) {
    error(__FUNCTION__, TensorError_InvalidAxis);
    return {};
  }

  const auto retShape = getReduceShape(t, dim, false);
  auto ret = TensorImpl::shape(retShape, t.device_);

  // first dim
  if (dim == 0) {
    const auto dimSize = t.shape_.front();
    if (static_cast<uint32_t>(dimSize) < getBlockSize()) {
      kReduceDimFirstOrLast<OP, true>
          <<<getGridSize(t.elemCount_), getBlockSize()>>>(
              ret.data_, t.data_, dimSize, ret.elemCount_);
      CUDA_KERNEL_CHECK();
    } else {
      reduceDimFirst<OpCudaReduceSum>(ret.data_, t.data_, dimSize,
                                      ret.elemCount_);
    }
  } else if (dim == t.dimCount_ - 1) {
    // last dim
    const auto dimSize = t.shape_.back();
    if (static_cast<uint32_t>(dimSize) < getBlockSize()) {
      kReduceDimFirstOrLast<OP, false>
          <<<getGridSize(t.elemCount_), getBlockSize()>>>(
              ret.data_, t.data_, dimSize, ret.elemCount_);
      CUDA_KERNEL_CHECK();
    } else {
      reduceDimLast<OpCudaReduceSum>(ret.data_, t.data_, dimSize,
                                     ret.elemCount_);
    }
  } else {
    // other dim
    auto ctxT = getTensorCtx(t);
    auto ctxRet = getTensorCtx(ret);
    kReduceDim<OP><<<getGridSize(t.elemCount_), getBlockSize()>>>(
        ctxRet, ctxT, dim, ctxRet.elemCount_);
    CUDA_KERNEL_CHECK();
  }

  if (keepDims) {
    const auto shapeKeepDims = getReduceShape(t, dim, true);
    ret.reshape_(shapeKeepDims);
  }
  return ret;
}

template <typename OP>
std::pair<TensorImpl, TensorImpl> TensorOpsCUDA::reduceIdxDim(
    const TensorImpl& t, int32_t dim, bool keepDims) {
  if (dim < 0) {
    dim += t.dimCount_;
  }
  if (dim < 0 || dim >= t.dimCount_) {
    error(__FUNCTION__, TensorError_InvalidAxis);
    return {};
  }

  const auto retShape = getReduceShape(t, dim, false);
  auto values = TensorImpl::shape(retShape, t.device_);
  auto indices = TensorImpl::shape(retShape, t.device_);

  if (dim == 0) {
    // first dim
    const auto dimSize = t.shape_.front();
    if (static_cast<uint32_t>(dimSize) < getBlockSize()) {
      kReduceIdxDimFirstOrLast<OP, true>
          <<<getGridSize(t.elemCount_), getBlockSize()>>>(
              values.data_, indices.data_, t.data_, dimSize, values.elemCount_);
      CUDA_KERNEL_CHECK();
    } else {
      reduceIdxDimFirst<OP>(values.data_, indices.data_, t.data_, dimSize,
                            values.elemCount_);
    }
  } else if (dim == t.dimCount_ - 1) {
    // last dim
    const auto dimSize = t.shape_.back();
    if (static_cast<uint32_t>(dimSize) < getBlockSize()) {
      kReduceIdxDimFirstOrLast<OP, false>
          <<<getGridSize(t.elemCount_), getBlockSize()>>>(
              values.data_, indices.data_, t.data_, dimSize, values.elemCount_);
      CUDA_KERNEL_CHECK();
    } else {
      reduceIdxDimLast<OP>(values.data_, indices.data_, t.data_, dimSize,
                           values.elemCount_);
    }
  } else {
    // other dim
    auto ctxT = getTensorCtx(t);
    auto ctxValues = getTensorCtx(values);
    kReduceIdxDim<OP><<<getGridSize(t.elemCount_), getBlockSize()>>>(
        ctxValues, indices.data_, ctxT, dim, ctxValues.elemCount_);
    CUDA_KERNEL_CHECK();
  }

  if (keepDims) {
    const auto shapeKeepDims = getReduceShape(t, dim, true);
    values.reshape_(shapeKeepDims);
    indices.reshape_(shapeKeepDims);
  }
  return {values, indices};
}

void TensorOpsCUDA::transpose2D(float* out, const float* in, int32_t width,
                                int32_t height) {
  dim3 blockSize(TRANSPOSE_TILE_DIM, TRANSPOSE_TILE_DIM);
  dim3 gridSize((width + TRANSPOSE_TILE_DIM - 1) / TRANSPOSE_TILE_DIM,
                (height + TRANSPOSE_TILE_DIM - 1) / TRANSPOSE_TILE_DIM);
  kTranspose<<<gridSize, blockSize>>>(out, in, width, height);
  CUDA_KERNEL_CHECK();
}

void TensorOpsCUDA::allocate(void** ptr, size_t size) {
  allocator_.allocate(ptr, size);
}

void TensorOpsCUDA::deallocate(void* ptr) { allocator_.deallocate(ptr); }

void TensorOpsCUDA::copyHostToDevice(void* dst, const void* src, size_t count) {
  CUDA_CHECK(hipMemcpy(dst, src, count, hipMemcpyHostToDevice));
}

void TensorOpsCUDA::copyOnDevice(void* dst, const void* src, size_t count) {
  CUDA_CHECK(hipMemcpy(dst, src, count, hipMemcpyDeviceToDevice));
}

void TensorOpsCUDA::copyDeviceToHost(void* dst, const void* src, size_t count) {
  CUDA_CHECK(hipMemcpy(dst, src, count, hipMemcpyDeviceToHost));
}

void TensorOpsCUDA::fillConstant_(float* dst, float val, size_t count) {
  kFillConstant<<<getGridSize(count, 4), getBlockSize()>>>(
      dst, val, static_cast<int32_t>(count));
  CUDA_KERNEL_CHECK();
}

void TensorOpsCUDA::fillConstant_(TensorImpl& t, float val) {
  kFillConstant<<<getGridSize(t.elemCount_, 4), getBlockSize()>>>(t.data_, val,
                                                                  t.elemCount_);
  CUDA_KERNEL_CHECK();
}

void TensorOpsCUDA::fillLinSpace_(float* dst, float start, float step,
                                  size_t count) {
  kFillLinSpace<<<getGridSize(count, 4), getBlockSize()>>>(
      dst, start, step, static_cast<int32_t>(count));
  CUDA_KERNEL_CHECK();
}

void TensorOpsCUDA::fillRandUniform_(TensorImpl& t, float min, float max) {
  auto seed = RandomGeneratorCUDA::getSeed();
  auto seq = RandomGeneratorCUDA::nextSequence();
  kFillRandUniform<<<getGridSize(t.elemCount_, 4), getBlockSize()>>>(
      t.data_, min, max, seed, seq, t.elemCount_);
  CUDA_KERNEL_CHECK();
}

void TensorOpsCUDA::fillRandNormal_(TensorImpl& t) {
  auto seed = RandomGeneratorCUDA::getSeed();
  auto seq = RandomGeneratorCUDA::nextSequence();
  kFillRandNormal<<<getGridSize(t.elemCount_, 4), getBlockSize()>>>(
      t.data_, 0.f, 1.f, seed, seq, t.elemCount_);
  CUDA_KERNEL_CHECK();
}

void TensorOpsCUDA::fillRandBernoulli_(TensorImpl& t, float p) {
  auto seed = RandomGeneratorCUDA::getSeed();
  auto seq = RandomGeneratorCUDA::nextSequence();
  kFillRandBernoulli<<<getGridSize(t.elemCount_, 4), getBlockSize()>>>(
      t.data_, p, seed, seq, t.elemCount_);
  CUDA_KERNEL_CHECK();
}

TensorImpl TensorOpsCUDA::add(const TensorImpl& a, const TensorImpl& b) {
  if (a.dimCount_ == 0) {
    return opPairScalarFirst<OpCudaAdd>(a, b);
  }
  if (b.dimCount_ == 0) {
    return opPairScalarSecond<OpCudaAdd>(a, b);
  }
  return opPairBroadcast<OpCudaAdd>(a, b);
}

TensorImpl TensorOpsCUDA::sub(const TensorImpl& a, const TensorImpl& b) {
  if (a.dimCount_ == 0) {
    return opPairScalarFirst<OpCudaSub>(a, b);
  }
  if (b.dimCount_ == 0) {
    return opPairScalarSecond<OpCudaSub>(a, b);
  }
  return opPairBroadcast<OpCudaSub>(a, b);
}

TensorImpl TensorOpsCUDA::mul(const TensorImpl& a, const TensorImpl& b) {
  if (a.dimCount_ == 0) {
    return opPairScalarFirst<OpCudaMul>(a, b);
  }
  if (b.dimCount_ == 0) {
    return opPairScalarSecond<OpCudaMul>(a, b);
  }
  return opPairBroadcast<OpCudaMul>(a, b);
}

TensorImpl TensorOpsCUDA::div(const TensorImpl& a, const TensorImpl& b) {
  if (a.dimCount_ == 0) {
    return opPairScalarFirst<OpCudaDiv>(a, b);
  }
  if (b.dimCount_ == 0) {
    return opPairScalarSecond<OpCudaDiv>(a, b);
  }
  return opPairBroadcast<OpCudaDiv>(a, b);
}

TensorImpl TensorOpsCUDA::pow(const TensorImpl& a, const TensorImpl& b) {
  if (b.dimCount_ == 0) {
    return opPairScalarSecond<OpCudaPow>(a, b);
  }
  return opPairBroadcast<OpCudaPow>(a, b);
}

TensorImpl TensorOpsCUDA::add(const TensorImpl& a, const float& b) {
  return opPair<OpCudaAdd>(a, b);
}

TensorImpl TensorOpsCUDA::sub(const TensorImpl& a, const float& b) {
  return opPair<OpCudaSub>(a, b);
}

TensorImpl TensorOpsCUDA::mul(const TensorImpl& a, const float& b) {
  return opPair<OpCudaMul>(a, b);
}

TensorImpl TensorOpsCUDA::div(const TensorImpl& a, const float& b) {
  return opPair<OpCudaMul>(a, 1.f / b);
}

TensorImpl TensorOpsCUDA::pow(const TensorImpl& a, const float& b) {
  return opPair<OpCudaPow>(a, b);
}

TensorImpl TensorOpsCUDA::add(const float& a, const TensorImpl& b) {
  return opPair<OpCudaAdd>(a, b);
}

TensorImpl TensorOpsCUDA::sub(const float& a, const TensorImpl& b) {
  return opPair<OpCudaSub>(a, b);
}

TensorImpl TensorOpsCUDA::mul(const float& a, const TensorImpl& b) {
  return opPair<OpCudaMul>(a, b);
}

TensorImpl TensorOpsCUDA::div(const float& a, const TensorImpl& b) {
  return opPair<OpCudaDiv>(a, b);
}

void TensorOpsCUDA::add_(TensorImpl& a, const TensorImpl& b) {
  if (a.dimCount_ == 0) {
    opPairScalarFirst_<OpCudaAdd>(a, b);
    return;
  }
  if (b.dimCount_ == 0) {
    opPairScalarSecond_<OpCudaAdd>(a, b);
    return;
  }
  opPairBroadcast_<OpCudaAdd>(a, b);
}

void TensorOpsCUDA::sub_(TensorImpl& a, const TensorImpl& b) {
  if (a.dimCount_ == 0) {
    opPairScalarFirst_<OpCudaSub>(a, b);
    return;
  }
  if (b.dimCount_ == 0) {
    opPairScalarSecond_<OpCudaSub>(a, b);
    return;
  }
  opPairBroadcast_<OpCudaSub>(a, b);
}

void TensorOpsCUDA::mul_(TensorImpl& a, const TensorImpl& b) {
  if (a.dimCount_ == 0) {
    opPairScalarFirst_<OpCudaMul>(a, b);
    return;
  }
  if (b.dimCount_ == 0) {
    opPairScalarSecond_<OpCudaMul>(a, b);
    return;
  }
  opPairBroadcast_<OpCudaMul>(a, b);
}

void TensorOpsCUDA::div_(TensorImpl& a, const TensorImpl& b) {
  if (a.dimCount_ == 0) {
    opPairScalarFirst_<OpCudaDiv>(a, b);
    return;
  }
  if (b.dimCount_ == 0) {
    opPairScalarSecond_<OpCudaDiv>(a, b);
    return;
  }
  opPairBroadcast_<OpCudaDiv>(a, b);
}

void TensorOpsCUDA::add_(TensorImpl& a, const float& b) {
  opPair_<OpCudaAdd>(a, b);
}

void TensorOpsCUDA::sub_(TensorImpl& a, const float& b) {
  opPair_<OpCudaSub>(a, b);
}

void TensorOpsCUDA::mul_(TensorImpl& a, const float& b) {
  opPair_<OpCudaMul>(a, b);
}

void TensorOpsCUDA::div_(TensorImpl& a, const float& b) {
  opPair_<OpCudaMul>(a, 1.f / b);
}

TensorImpl TensorOpsCUDA::eq(const TensorImpl& a, const TensorImpl& b) {
  if (b.dimCount_ == 0) {
    return opPairScalarSecond<OpCudaEq>(a, b);
  }
  return opPairBroadcast<OpCudaEq>(a, b);
}

TensorImpl TensorOpsCUDA::ne(const TensorImpl& a, const TensorImpl& b) {
  if (b.dimCount_ == 0) {
    return opPairScalarSecond<OpCudaNe>(a, b);
  }
  return opPairBroadcast<OpCudaNe>(a, b);
}

TensorImpl TensorOpsCUDA::ge(const TensorImpl& a, const TensorImpl& b) {
  if (b.dimCount_ == 0) {
    return opPairScalarSecond<OpCudaGe>(a, b);
  }
  return opPairBroadcast<OpCudaGe>(a, b);
}

TensorImpl TensorOpsCUDA::gt(const TensorImpl& a, const TensorImpl& b) {
  if (b.dimCount_ == 0) {
    return opPairScalarSecond<OpCudaGt>(a, b);
  }
  return opPairBroadcast<OpCudaGt>(a, b);
}

TensorImpl TensorOpsCUDA::le(const TensorImpl& a, const TensorImpl& b) {
  if (b.dimCount_ == 0) {
    return opPairScalarSecond<OpCudaLe>(a, b);
  }
  return opPairBroadcast<OpCudaLe>(a, b);
}

TensorImpl TensorOpsCUDA::lt(const TensorImpl& a, const TensorImpl& b) {
  if (b.dimCount_ == 0) {
    return opPairScalarSecond<OpCudaLt>(a, b);
  }
  return opPairBroadcast<OpCudaLt>(a, b);
}

TensorImpl TensorOpsCUDA::maximum(const TensorImpl& a, const TensorImpl& b) {
  if (b.dimCount_ == 0) {
    return opPairScalarSecond<OpCudaMax>(a, b);
  }
  return opPairBroadcast<OpCudaMax>(a, b);
}

TensorImpl TensorOpsCUDA::minimum(const TensorImpl& a, const TensorImpl& b) {
  if (b.dimCount_ == 0) {
    return opPairScalarSecond<OpCudaMin>(a, b);
  }
  return opPairBroadcast<OpCudaMin>(a, b);
}

TensorImpl TensorOpsCUDA::eq(const TensorImpl& a, const float& b) {
  return opPair<OpCudaEq>(a, b);
}

TensorImpl TensorOpsCUDA::ne(const TensorImpl& a, const float& b) {
  return opPair<OpCudaNe>(a, b);
}

TensorImpl TensorOpsCUDA::ge(const TensorImpl& a, const float& b) {
  return opPair<OpCudaGe>(a, b);
}

TensorImpl TensorOpsCUDA::gt(const TensorImpl& a, const float& b) {
  return opPair<OpCudaGt>(a, b);
}

TensorImpl TensorOpsCUDA::le(const TensorImpl& a, const float& b) {
  return opPair<OpCudaLe>(a, b);
}

TensorImpl TensorOpsCUDA::lt(const TensorImpl& a, const float& b) {
  return opPair<OpCudaLt>(a, b);
}

TensorImpl TensorOpsCUDA::maximum(const TensorImpl& a, const float& b) {
  return opPair<OpCudaMax>(a, b);
}

TensorImpl TensorOpsCUDA::minimum(const TensorImpl& a, const float& b) {
  return opPair<OpCudaMin>(a, b);
}

void TensorOpsCUDA::sin_(TensorImpl& t) { opSingle_<OpCudaSin_>(t); }

void TensorOpsCUDA::cos_(TensorImpl& t) { opSingle_<OpCudaCos_>(t); }

void TensorOpsCUDA::sqrt_(TensorImpl& t) { opSingle_<OpCudaSqrt_>(t); }

void TensorOpsCUDA::tanh_(TensorImpl& t) { opSingle_<OpCudaTanh_>(t); }

void TensorOpsCUDA::exp_(TensorImpl& t) { opSingle_<OpCudaExp_>(t); }

void TensorOpsCUDA::log_(TensorImpl& t) { opSingle_<OpCudaLog_>(t); }

TensorImpl TensorOpsCUDA::sin(const TensorImpl& t) {
  return opSingle<OpCudaSin>(t);
}

TensorImpl TensorOpsCUDA::cos(const TensorImpl& t) {
  return opSingle<OpCudaCos>(t);
}

TensorImpl TensorOpsCUDA::sqrt(const TensorImpl& t) {
  return opSingle<OpCudaSqrt>(t);
}

TensorImpl TensorOpsCUDA::tanh(const TensorImpl& t) {
  return opSingle<OpCudaTanh>(t);
}

TensorImpl TensorOpsCUDA::exp(const TensorImpl& t) {
  return opSingle<OpCudaExp>(t);
}

TensorImpl TensorOpsCUDA::log(const TensorImpl& t) {
  return opSingle<OpCudaLog>(t);
}

void TensorOpsCUDA::clampMin_(TensorImpl& t, float min) {
  opPair_<OpCudaMax>(t, min);
}

void TensorOpsCUDA::clampMax_(TensorImpl& t, float max) {
  opPair_<OpCudaMin>(t, max);
}

void TensorOpsCUDA::clamp_(TensorImpl& t, float min, float max) {
  kClamp_<<<getGridSize(t.elemCount_), getBlockSize()>>>(t.data_, min, max,
                                                         t.elemCount_);
  CUDA_KERNEL_CHECK();
}

TensorImpl TensorOpsCUDA::clampMin(const TensorImpl& t, float min) {
  return opPair<OpCudaMax>(t, min);
}

TensorImpl TensorOpsCUDA::clampMax(const TensorImpl& t, float max) {
  return opPair<OpCudaMin>(t, max);
}

TensorImpl TensorOpsCUDA::clamp(const TensorImpl& t, float min, float max) {
  auto ret = TensorImpl::shape(t.shape_, t.device_);
  kClamp<<<getGridSize(t.elemCount_), getBlockSize()>>>(ret.data_, t.data_, min,
                                                        max, t.elemCount_);
  CUDA_KERNEL_CHECK();
  return ret;
}

TensorImpl TensorOpsCUDA::min(const TensorImpl& t) {
  if (t.dimCount_ == 0) {
    return t;
  }
  auto ret = TensorImpl::scalar(t.device_);
  reduceMerge<OpCudaReduceMin, ReduceIndexAll>(ret.data_, t.data_,
                                               t.elemCount_);
  return ret;
}

TensorImpl TensorOpsCUDA::max(const TensorImpl& t) {
  if (t.dimCount_ == 0) {
    return t;
  }
  auto ret = TensorImpl::scalar(t.device_);
  reduceMerge<OpCudaReduceMax, ReduceIndexAll>(ret.data_, t.data_,
                                               t.elemCount_);
  return ret;
}

TensorImpl TensorOpsCUDA::argmin(const TensorImpl& t) {
  if (t.dimCount_ == 0) {
    return TensorImpl::scalar(0, t.device_);
  }
  auto ret = TensorImpl::scalar(t.device_);
  reduceIdxMerge<OpCudaReduceMin, ReduceIndexAll>(nullptr, ret.data_, t.data_,
                                                  t.elemCount_);
  return ret;
}

TensorImpl TensorOpsCUDA::argmax(const TensorImpl& t) {
  if (t.dimCount_ == 0) {
    return TensorImpl::scalar(0, t.device_);
  }
  auto ret = TensorImpl::scalar(t.device_);
  reduceIdxMerge<OpCudaReduceMax, ReduceIndexAll>(nullptr, ret.data_, t.data_,
                                                  t.elemCount_);
  return ret;
}

TensorImpl TensorOpsCUDA::sum(const TensorImpl& t) {
  if (t.dimCount_ == 0) {
    return t;
  }
  auto ret = TensorImpl::scalar(t.device_);
  reduceMerge<OpCudaReduceSum, ReduceIndexAll>(ret.data_, t.data_,
                                               t.elemCount_);
  return ret;
}

TensorImpl TensorOpsCUDA::mean(const TensorImpl& t) {
  if (t.dimCount_ == 0) {
    return t;
  }
  auto ret = TensorImpl::scalar(t.device_);
  reduceMerge<OpCudaReduceSum, ReduceIndexAll>(ret.data_, t.data_,
                                               t.elemCount_);
  const auto r = 1.f / static_cast<float>(t.elemCount_);
  mul_(ret, r);
  return ret;
}

std::pair<TensorImpl, TensorImpl> TensorOpsCUDA::varMean(const TensorImpl& t,
                                                         bool unbiased) {
  if (t.dimCount_ == 0) {
    return {TensorImpl::scalar(0, t.device_), t};
  }
  const auto meanVal = mean(t);
  const auto squaredDiff = TensorImpl::shape({t.elemCount_}, t.device_);
  kSquaredDiff<<<getGridSize(t.elemCount_), getBlockSize()>>>(
      squaredDiff.data_, t.data_, meanVal.data_, t.elemCount_);

  auto varVal = TensorImpl::scalar(t.device_);
  reduceMerge<OpCudaReduceSum, ReduceIndexAll>(varVal.data_, squaredDiff.data_,
                                               t.elemCount_);

  const auto n = static_cast<float>(t.elemCount_);
  auto r = 1.f / n;
  if (unbiased) {
    r *= n / (n - 1.f);
  }
  mul_(varVal, r);
  return {varVal, meanVal};
}

std::pair<TensorImpl, TensorImpl> TensorOpsCUDA::min(const TensorImpl& t,
                                                     int32_t dim,
                                                     bool keepDims) {
  if (t.dimCount_ == 0) {
    return {t, TensorImpl::scalar(0, t.device_)};
  }
  return reduceIdxDim<OpCudaReduceMin>(t, dim, keepDims);
}

std::pair<TensorImpl, TensorImpl> TensorOpsCUDA::max(const TensorImpl& t,
                                                     int32_t dim,
                                                     bool keepDims) {
  if (t.dimCount_ == 0) {
    return {t, TensorImpl::scalar(0, t.device_)};
  }
  return reduceIdxDim<OpCudaReduceMax>(t, dim, keepDims);
}

TensorImpl TensorOpsCUDA::sum(const TensorImpl& t,
                              const std::vector<int32_t>& dims, bool keepDims) {
  if (t.dimCount_ == 0) {
    return t;
  }
  if (dims.size() == 1) {
    return reduceDim<OpCudaReduceSum>(t, dims[0], keepDims);
  }

  FixedVector<uint8_t> inAxis{};
  for (int32_t d : dims) {
    if (d < 0) {
      d += t.dimCount_;
    }
    if (d < 0 || d >= t.dimCount_) {
      error(__FUNCTION__, TensorError_InvalidAxis);
      return {};
    }
    inAxis.data[d] = 1;
  }

  auto retShape = getReduceShape(t, inAxis, keepDims);
  auto ret = TensorImpl::shape(retShape, t.device_);

  auto ctxT = getTensorCtx(t);
  fillConstant_(ret, 0);
  kReduceMultiDimSum<<<getGridSize(t.elemCount_), getBlockSize()>>>(
      ret.data_, ctxT, inAxis, t.elemCount_);
  CUDA_KERNEL_CHECK();
  return ret;
}

TensorImpl TensorOpsCUDA::mean(const TensorImpl& t,
                               const std::vector<int32_t>& dims,
                               bool keepDims) {
  auto ret = sum(t, dims, keepDims);
  if (!ret.empty()) {
    auto r =
        static_cast<float>(ret.elemCount_) / static_cast<float>(t.elemCount_);
    mul_(ret, r);
  }
  return ret;
}

std::pair<TensorImpl, TensorImpl> TensorOpsCUDA::varMean(
    const TensorImpl& t, const std::vector<int32_t>& dims, bool unbiased,
    bool keepDims) {
  if (t.dimCount_ == 0) {
    return {TensorImpl::scalar(0, t.device_), t};
  }
  FixedVector<uint8_t> inAxis{};
  for (int32_t d : dims) {
    if (d < 0) {
      d += t.dimCount_;
    }
    if (d < 0 || d >= t.dimCount_) {
      error(__FUNCTION__, TensorError_InvalidAxis);
      return {};
    }
    inAxis.data[d] = 1;
  }

  auto meanVal = mean(t, dims, true);

  auto retShape = getReduceShape(t, inAxis, keepDims);
  auto varVal = TensorImpl::shape(retShape, t.device_);
  fillConstant_(varVal, 0);

  auto ctxT = getTensorCtx(t);
  kReduceMultiDimVar<<<getGridSize(t.elemCount_), getBlockSize()>>>(
      varVal.data_, ctxT, meanVal.data_, inAxis, t.elemCount_);
  CUDA_KERNEL_CHECK();

  auto reduceSize =
      static_cast<float>(t.elemCount_) / static_cast<float>(varVal.elemCount_);
  auto r = 1.f / reduceSize;
  if (unbiased) {
    r *= reduceSize / (reduceSize - 1.f);
  }
  mul_(varVal, r);
  return {varVal, meanVal};
}

TensorImpl TensorOpsCUDA::permute(const TensorImpl& t,
                                  const std::vector<int32_t>& dims) {
  auto retShape = t.shape_;
  reorderIndices(retShape.data(), dims);
  auto ret = TensorImpl::shape(retShape, t.device_);

  auto ctxT = getTensorCtx(t);
  auto ctxRet = getTensorCtx(ret);

  FixedVector<int32_t> dimsData;
  for (auto i = 0; i < t.dimCount_; i++) {
    dimsData.data[i] = dims[i];
  }
  kPermute<<<getGridSize(t.elemCount_), getBlockSize()>>>(
      ctxRet, ctxT, dimsData, t.elemCount_);
  CUDA_KERNEL_CHECK();
  return ret;
}

TensorImpl TensorOpsCUDA::transpose2D(const TensorImpl& t) {
  auto ret = TensorImpl::shape({t.shape_[1], t.shape_[0]}, t.device_);
  transpose2D(ret.data_, t.data_, t.shape_[1], t.shape_[0]);
  return ret;
}

TensorImpl TensorOpsCUDA::index(
    const TensorImpl& t,
    const std::vector<std::reference_wrapper<TensorImpl>>& indices) {
  auto len = static_cast<int32_t>(indices.size());
  auto firstDim = indices[0].get().elemCount_;
  auto dimStride = t.strides_[len - 1];
  Shape retShape = {firstDim};
  for (auto i = len; i < t.dimCount_; i++) {
    retShape.push_back(t.shape_[i]);
  }
  auto retTensor = TensorImpl::shape(retShape, t.device_);

  // 2D
  if (t.dimCount_ == 2 && len == 2) {
    kIndex2D<<<getGridSize(firstDim), getBlockSize()>>>(
        retTensor.data_, t.data_, indices[0].get().data_,
        indices[1].get().data_, t.shape_[0], t.shape_[1], firstDim);
    CUDA_KERNEL_CHECK();
    return retTensor;
  }

  FixedVector<float*> indicesData{};
  for (int32_t i = 0; i < len; i++) {
    indicesData.data[i] = indices[i].get().data_;
  }
  auto ctxT = getTensorCtx(t);
  kIndex<<<getGridSize(firstDim), getBlockSize()>>>(
      retTensor.data_, ctxT, indicesData, dimStride, len, firstDim);
  CUDA_KERNEL_CHECK();
  return retTensor;
}

void TensorOpsCUDA::indexPut_(
    TensorImpl& t,
    const std::vector<std::reference_wrapper<TensorImpl>>& indices, float val) {
  auto len = static_cast<int32_t>(indices.size());
  auto firstDim = indices[0].get().elemCount_;
  auto dimStride = t.strides_[len - 1];

  // 2D
  if (t.dimCount_ == 2 && len == 2) {
    kIndexPut2D<<<getGridSize(firstDim), getBlockSize()>>>(
        t.data_, indices[0].get().data_, indices[1].get().data_, t.shape_[0],
        t.shape_[1], val, firstDim);
    CUDA_KERNEL_CHECK();
    return;
  }

  FixedVector<float*> indicesData{};
  for (int32_t i = 0; i < len; i++) {
    indicesData.data[i] = indices[i].get().data_;
  }
  auto ctxT = getTensorCtx(t);
  kIndexPut<<<getGridSize(firstDim), getBlockSize()>>>(
      ctxT, indicesData, dimStride, len, val, firstDim);
  CUDA_KERNEL_CHECK();
}

void TensorOpsCUDA::indexPut_(
    TensorImpl& t,
    const std::vector<std::reference_wrapper<TensorImpl>>& indices,
    const TensorImpl& val) {
  auto len = static_cast<int32_t>(indices.size());
  auto firstDim = indices[0].get().elemCount_;
  auto dimStride = t.strides_[len - 1];
  assert(val.elemCount_ == dimStride * firstDim);

  // 2D
  if (t.dimCount_ == 2 && len == 2) {
    kIndexPut2D<<<getGridSize(firstDim), getBlockSize()>>>(
        t.data_, indices[0].get().data_, indices[1].get().data_, t.shape_[0],
        t.shape_[1], val.data_, firstDim);
    CUDA_KERNEL_CHECK();
    return;
  }

  FixedVector<float*> indicesData{};
  for (int32_t i = 0; i < len; i++) {
    indicesData.data[i] = indices[i].get().data_;
  }
  auto ctxT = getTensorCtx(t);
  kIndexPut<<<getGridSize(firstDim), getBlockSize()>>>(
      ctxT, indicesData, dimStride, len, val.data_, firstDim);
  CUDA_KERNEL_CHECK();
}

TensorImpl TensorOpsCUDA::im2col(const TensorImpl& t, Size2D kernel,
                                 Size2D stride, Size2D padding) {
  // shape: [C, H, W], [N, C, H, W]
  assert(t.dimCount_ == 3 || t.dimCount_ == 4);
  int32_t batch = (t.dimCount_ == 4) ? t.shape_[0] : 1;
  int32_t channels = (t.dimCount_ == 4) ? t.shape_[1] : t.shape_[0];
  int32_t height = (t.dimCount_ == 4) ? t.shape_[2] : t.shape_[1];
  int32_t width = (t.dimCount_ == 4) ? t.shape_[3] : t.shape_[2];
  int32_t outH = (height - kernel.h + 2 * padding.h) / stride.h + 1;
  int32_t outW = (width - kernel.w + 2 * padding.w) / stride.w + 1;

  int32_t colH = outH * outW;
  int32_t colW = channels * kernel.h * kernel.w;
  auto ret = TensorImpl::shape({batch * colH, colW}, t.device_);

  int32_t n = ret.elemCount_;
  kIm2Col<<<getGridSize(n), getBlockSize()>>>(
      ret.data_, t.data_, n, channels, height, width, outH, outW, kernel.h,
      kernel.w, stride.h, stride.w, padding.h, padding.w);
  CUDA_KERNEL_CHECK();
  return ret;
}

TensorImpl TensorOpsCUDA::col2im(const TensorImpl& t, const Shape& shape,
                                 Size2D kernel, Size2D stride, Size2D padding) {
  // shape: [C, H, W], [N, C, H, W]
  assert(shape.size() == 3 || shape.size() == 4);
  int32_t batch = (shape.size() == 4) ? shape[0] : 1;
  int32_t channels = (shape.size() == 4) ? shape[1] : shape[0];
  int32_t height = (shape.size() == 4) ? shape[2] : shape[1];
  int32_t width = (shape.size() == 4) ? shape[3] : shape[2];

  auto outH = (height - kernel.h + 2 * padding.h) / stride.h + 1;
  auto outW = (width - kernel.w + 2 * padding.w) / stride.w + 1;

  // int32_t colH = outH * outW;
  // int32_t colW = channels * kernel.h * kernel.w;
  auto ret = TensorImpl::zeros(shape, t.device_);

  int32_t n = batch * channels * outH * outW;
  kCol2Im<<<getGridSize(n), getBlockSize()>>>(
      ret.data_, t.data_, n, channels, height, width, outH, outW, kernel.h,
      kernel.w, stride.h, stride.w, padding.h, padding.w);
  CUDA_KERNEL_CHECK();
  return ret;
}

TensorImpl TensorOpsCUDA::dot(const TensorImpl& a, const TensorImpl& b) {
  auto ret = TensorImpl::scalar(0.f, a.device_);
  auto sharedMemSize = getBlockSize() * sizeof(float);
  kDot<<<getGridSize(a.elemCount_), getBlockSize(), sharedMemSize>>>(
      ret.data_, a.data_, b.data_, a.elemCount_);
  CUDA_KERNEL_CHECK();
  return ret;
}

void TensorOpsCUDA::gemm(float* c, const float* a, const float* b, int32_t m,
                         int32_t k, int32_t n, bool transA, bool transB) {
  hipblasOperation_t opA = transA ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  hipblasOperation_t opB = transB ? HIPBLAS_OP_T : HIPBLAS_OP_N;

  int lda = transA ? m : k;
  int ldb = transB ? k : n;
  int ldc = n;

  constexpr float alpha = 1.f;
  constexpr float beta = 0.f;

  CUBLAS_CHECK(hipblasSgemm(getCublasHandle(), opB, opA, n, m, k, &alpha, b, ldb,
                           a, lda, &beta, c, ldc));
}

const char* curandGetErrorString(hiprandStatus_t status) {
  switch (status) {
    case HIPRAND_STATUS_SUCCESS:
      return "HIPRAND_STATUS_SUCCESS";
    case HIPRAND_STATUS_VERSION_MISMATCH:
      return "HIPRAND_STATUS_VERSION_MISMATCH";
    case HIPRAND_STATUS_NOT_INITIALIZED:
      return "HIPRAND_STATUS_NOT_INITIALIZED";
    case HIPRAND_STATUS_ALLOCATION_FAILED:
      return "HIPRAND_STATUS_ALLOCATION_FAILED";
    case HIPRAND_STATUS_TYPE_ERROR:
      return "HIPRAND_STATUS_TYPE_ERROR";
    case HIPRAND_STATUS_OUT_OF_RANGE:
      return "HIPRAND_STATUS_OUT_OF_RANGE";
    case HIPRAND_STATUS_LENGTH_NOT_MULTIPLE:
      return "HIPRAND_STATUS_LENGTH_NOT_MULTIPLE";
    case HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED:
      return "HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED";
    case HIPRAND_STATUS_LAUNCH_FAILURE:
      return "HIPRAND_STATUS_LAUNCH_FAILURE";
    case HIPRAND_STATUS_PREEXISTING_FAILURE:
      return "HIPRAND_STATUS_PREEXISTING_FAILURE";
    case HIPRAND_STATUS_INITIALIZATION_FAILED:
      return "HIPRAND_STATUS_INITIALIZATION_FAILED";
    case HIPRAND_STATUS_ARCH_MISMATCH:
      return "HIPRAND_STATUS_ARCH_MISMATCH";
    case HIPRAND_STATUS_INTERNAL_ERROR:
      return "HIPRAND_STATUS_INTERNAL_ERROR";
  }
  return "Unknown cuRAND error";
}

const char* cublasGetErrorString(hipblasStatus_t status) {
  switch (status) {
    case HIPBLAS_STATUS_SUCCESS:
      return "HIPBLAS_STATUS_SUCCESS";
    case HIPBLAS_STATUS_NOT_INITIALIZED:
      return "HIPBLAS_STATUS_NOT_INITIALIZED";
    case HIPBLAS_STATUS_ALLOC_FAILED:
      return "HIPBLAS_STATUS_ALLOC_FAILED";
    case HIPBLAS_STATUS_INVALID_VALUE:
      return "HIPBLAS_STATUS_INVALID_VALUE";
    case HIPBLAS_STATUS_ARCH_MISMATCH:
      return "HIPBLAS_STATUS_ARCH_MISMATCH";
    case HIPBLAS_STATUS_MAPPING_ERROR:
      return "HIPBLAS_STATUS_MAPPING_ERROR";
    case HIPBLAS_STATUS_EXECUTION_FAILED:
      return "HIPBLAS_STATUS_EXECUTION_FAILED";
    case HIPBLAS_STATUS_INTERNAL_ERROR:
      return "HIPBLAS_STATUS_INTERNAL_ERROR";
    case HIPBLAS_STATUS_NOT_SUPPORTED:
      return "HIPBLAS_STATUS_NOT_SUPPORTED";
    case HIPBLAS_STATUS_UNKNOWN:
      return "HIPBLAS_STATUS_UNKNOWN";
  }
  return "Unknown cuBLAS error";
}

}  // namespace TinyTorch
